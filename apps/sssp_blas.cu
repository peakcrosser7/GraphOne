#include "hip/hip_runtime.h"

#include <iostream>
#include <limits>
#include <chrono>

#include "GraphGenlX/graph_genlx.h"

using namespace std;
using namespace graph_genlx;

constexpr arch_t arch = arch_t::cuda;
using dist_t = float;

constexpr dist_t kMaxDist = std::numeric_limits<dist_t>::max();

struct sssp_hstatus_t {
    vid_t src_vid;

    DenseVec<arch, dist_t>& dists;
    // DenseVec<arch, uint32_t>& visited;
};

struct sssp_dstatus_t {
    uint32_t iter;
    dist_t * dists;
    // uint32_t* visited;
};

template <typename graph_t>
struct SSSPComp : ComponentX<graph_t, sssp_hstatus_t, sssp_dstatus_t> {
    using comp_t = ComponentX<graph_t, sssp_hstatus_t, sssp_dstatus_t>;
    using comp_t::ComponentX;

    void Init() override {
        auto& src_vid = this->h_status.src_vid;
        auto& dists = this->h_status.dists;

        archi::fill<arch>(dists.begin(), dists.end(), kMaxDist);
        dists.set(src_vid, 0);
    }

    void BeforeEngine() override {
        ++this->d_status.iter;
    }
};

struct SSSPFunctor : BlasFunctor<vid_t, dist_t, sssp_dstatus_t, dist_t, dist_t> {
    static dist_t default_info() {
        return kMaxDist;
    }

    __GENLX_ARCH_INL__
    static dist_t default_result() {
        return kMaxDist;
    }

    __GENLX_DEV_INL__
    static dist_t construct(const vid_t& vid, const sssp_dstatus_t& d_status) {
        return d_status.dists[vid];
    }

    __GENLX_DEV_INL__
    static dist_t gather(const dist_t& weight, const dist_t& info) {
        return (info == kMaxDist) ? info : weight + info;
    }

    __GENLX_DEV_INL__
    static dist_t reduce(const dist_t& lhs, const dist_t& rhs) {
        return std::min(lhs, rhs);
    }

    __GENLX_DEV_INL__
    static bool apply(const vid_t& vid, const dist_t& res, sssp_dstatus_t& d_status) {
        if (res < d_status.dists[vid]) {
            d_status.dists[vid] = res;
            return true;
        }
        return false;
    }

};

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printx("Usage: ", argv[0], " <graph_file> <src_vertex>");
        return -1;
    }
    
    vid_t src = std::stoi(argv[2]);

    Loader<vstart_t::FROM_1_TO_1, false> loader;
    LoadEdgeOpts opts;
    opts.comment_prefix = "%";
    // opts.is_directed = true;
    auto cache = loader.LoadEdgesFromTxt<dist_t>(argv[1], opts);
    auto g = graph::build<arch_t::cuda, BlasViews>(cache);
    using graph_t = decltype(g);

    if (!loader.ReorderedVid(src)) {
        LOG_ERROR("src vertex \"", src, "\" is not exist");
    }
    
    DenseVec<arch, dist_t> dists(g.num_vertices());
    DenseVec<arch, vid_t> visited(g.num_vertices());

    sssp_hstatus_t h_status{src, dists/*, visited*/};
    sssp_dstatus_t d_status{0, dists.data()/*, visited.data()*/};
    DblBufFrontier<arch, vid_t> frontier(g.num_vertices(), src);

    SSSPComp<graph_t> comp(g, h_status, d_status);

    auto start = std::chrono::high_resolution_clock::now();
    Run<SSSPFunctor>(comp, frontier);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    printx("Elapsed time: ", duration.count(), "ms");
 
    auto h_dists = dists.to<arch_t::cpu>();
    for (int i = 0; i < min(500, h_dists.size()); ++i) {
        printx(i, "-", h_dists[i]);
    }
    return 0;
}